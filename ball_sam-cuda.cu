
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <hiprand/hiprand_kernel.h>

//bin and samples
const int n_bins = 100;
const int n_points = 3000;

__global__ void generatePoints(int dim, int *hist, int n_bins, int n_points){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n_points) {
        hiprandState state; //cuda rand num generator
        hiprand_init(tid, 0, 0, &state); 
        
        double sum_squares = 0;
	    do {
    	    sum_squares = 0.0;
            for (int i = 0; i < dim; ++i) {
                double p = hiprand_uniform(&state) * 2 - 1; //rand num between -1 and 1
                sum_squares += p * p;
	            if(sum_squares > 1.0)
	       	        break;
            }
	    } while(sum_squares > 1.0); //validation

	    double distance = sqrtf(sum_squares);
        int bin = min((int)(distance * n_bins), n_bins - 1);
        atomicAdd(&hist[bin], 1);
        
    }
}

int main() {
    int *hist; //pointer to allow for easy mem access from host to device
    hipMallocManaged(&hist, n_bins * sizeof(int));

    for (int dim = 2; dim <= 16; ++dim) {
        //allocate memory
        hipMemset(hist, 0, n_bins * sizeof(int));
        int blockSize = 256; //thread count
        int numBlocks = (n_points + blockSize - 1) / blockSize; //ensure enough blocks are created
        generatePoints<<<numBlocks, blockSize>>>(dim, hist, n_bins, n_points);
        hipDeviceSynchronize(); //synchronize threads

        // Print results
        std::cout << "Dimension: " << dim << "\n";
        for (int i = 0; i < n_bins; ++i) {
            std::cout << (double)(hist[i]) / n_points << " ";
        }
        std::cout << "\n\n";
    }

    hipFree(hist);
    return 0;
}
